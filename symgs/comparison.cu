
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#define BLOCKN 1
#define THREADN 512

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }


double get_time() { // function to get the time of day in second
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, float **matrixDiagonal, const char *filename, int *num_rows, int *num_cols, int *num_vals){
    FILE *file = fopen(filename, "r");
    if (file == NULL){
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");

    //printf("Rows: %d, Columns:%d, NNZ:%d\n", *num_rows, *num_cols, *num_vals);
    int *row_ptr_t = (int *)malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *)malloc(*num_vals * sizeof(int));
    float *values_t = (float *)malloc(*num_vals * sizeof(float));
    float *matrixDiagonal_t = (float *)malloc(*num_rows * sizeof(float));
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *)malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++){
        row_occurances[i] = 0;
    }

    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF){
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        row_occurances[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++){
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);

    // Set the file position to the beginning of the file
    rewind(file);

    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++){
        col_ind_t[i] = -1;
    }

    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0, j = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF){
        row--;
        column--;

        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1){
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        if (row == column){
            matrixDiagonal_t[j] = value;
            j++;
        }
        i = 0;
    }
    fclose(file);
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
    *matrixDiagonal = matrixDiagonal_t;
}

// CPU implementation of SYMGS using CSR, DO NOT CHANGE THIS
void symgs_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal){

    // forward sweep
    for (int i = 0; i < num_rows; i++){
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++){
            sum -= values[j] * x[col_ind[j]];
        }

        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }

    // backward sweep
    for (int i = num_rows - 1; i >= 0; i--){
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++){
            sum -= values[j] * x[col_ind[j]];
        }
        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }
}

__global__ void symgs_csr_gpu(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal, float* x2, char* locks, char* changed){
    int start, end, i;
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    int chunk_size = (int) num_rows / (BLOCKN * THREADN);
    start = chunk_size * index;
    end = chunk_size * (index + 1);

    if(blockIdx.x == BLOCKN - 1 && threadIdx.x == THREADN - 1)
        end = num_rows;
    
    for(i = start; i < end; i++){
        *(locks + i) = 0;
        *(changed + i) = 0;
    }

    __syncthreads();

    char missed;
    do{
        missed = 0;
        for(i = start; i < end; i++){
            if(changed[i])
                continue;
            
            float sum = x[i];
            const int row_start = row_ptr[i];
            const int row_end = row_ptr[i + 1];
            float currentDiagonal = matrixDiagonal[i]; // Current diagonal value
    
            for (int j = row_start; j < row_end; j++){
                int index = col_ind[j];
                if(j > i){
                    // new value is not ready yet, try next iteration
                    if(locks[j] == 0){
                        missed = 1;
                        continue;
                    }

                    sum -= values[j] * x2[index];
                }
                else
                    sum -= values[j] * x[index];
                
            }
            sum += x[i] * currentDiagonal;
            x2[i] = sum / currentDiagonal;
            locks[i] = 1;
            changed[i] = 1;
        }
    } while (missed);


    do{
        missed = 0;
        for(i = end - 1; i >= start; i--){
            if(! changed[i])
                continue;
            
            float sum = x2[i];
            const int row_start = row_ptr[i];
            const int row_end = row_ptr[i + 1];
            float currentDiagonal = matrixDiagonal[i]; // Current diagonal value
    
            for (int j = row_start; j < row_end; j++){
                int index = col_ind[j];
                if(j < i){
                    // new value is not ready yet, try next iteration
                    if(locks[j] == 1){
                        missed = 1;
                        continue;
                    }

                    sum -= values[j] * x2[index];
                }
                else
                    sum -= values[j] * x[index];
                
            }
            sum += x[i] * currentDiagonal;
            x2[i] = sum / currentDiagonal;
            locks[i] = 2;
            changed[i] = 0;
        }
    } while (missed);
    __syncthreads();
}

int main(int argc, const char *argv[]){
    /* if (argc != 2){
        printf("Usage: ./exec matrix_file");
        return 0;
    } */
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    float *matrixDiagonal;
    
    const char *filename = argv[2];
    //printf("%s\n", filename);

    double start_cpu, end_cpu;
    double start_gpu, end_gpu;

    read_matrix(&row_ptr, &col_ind, &values, &matrixDiagonal, "kmer_V4a.mtx", &num_rows, &num_cols, &num_vals);
    float *x = (float *)malloc(num_rows * sizeof(float));
    float *xCopy = (float *)malloc(num_rows * sizeof(float));

    // Generate a random vector
    srand(time(NULL));
    int zeros = 0;
    for (int i = 0; i < num_rows; i++){
        x[i] = (rand() % 100) / (rand() % 100 + 1); // the number we use to divide cannot be 0, that's the reason of the +1
        xCopy[i] = x[i];
        if(x[i] == 0)
            zeros ++;
    }
    //printf("%d\n", zeros);
    
    // Compute in sw
    start_cpu = get_time();
    symgs_csr_sw(row_ptr, col_ind, values, num_rows, x, matrixDiagonal);
    end_cpu = get_time();

    // gpu part
    //printf("Before gpu\n");
    // allocate space
    int *dev_row_ptr, *dev_col_ind;
    float *dev_values, *dev_x, *dev_matrixDiagonal, *dev_x2;
    char *dev_locks, *dev_changed;
    CHECK(hipMalloc(&dev_row_ptr, (num_rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&dev_col_ind, num_vals * sizeof(int)));
    CHECK(hipMalloc(&dev_values, num_vals * sizeof(float)));
    CHECK(hipMalloc(&dev_x, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_matrixDiagonal, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_x2, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_locks, num_rows * sizeof(char)));
    CHECK(hipMalloc(&dev_changed, num_rows * sizeof(char)));
    printf("after gpu malloc\n");


    CHECK(hipMemcpy(dev_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_col_ind, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_values, values, num_vals * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_x, xCopy, num_rows * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_matrixDiagonal, matrixDiagonal, num_rows * sizeof(float), hipMemcpyHostToDevice));

    printf("After gpu memcpy\n");

    dim3 blocksPerGrid(BLOCKN, 1, 1);
    dim3 threadsPerBlock(THREADN, 1, 1);
    // compute in gpu
    start_gpu = get_time();
    
    symgs_csr_gpu<<<blocksPerGrid, threadsPerBlock>>>(
        dev_row_ptr,
        dev_col_ind,
        dev_values,
        num_rows,
        dev_x,
        dev_matrixDiagonal,
        dev_x2,
        dev_locks,
        dev_changed
    );
    CHECK_KERNELCALL();

    printf("After gpu kernerlcall\n");
    CHECK(hipDeviceSynchronize());

    end_gpu = get_time();

    CHECK(hipMemcpy(&xCopy, dev_x, sizeof(float), hipMemcpyDeviceToHost));
    /* for(int i = 0; i< 100; i++)
        printf("%lf\n", x[i]); */


    printf("After gpu output memcpy\n");

    for(int i = 0; i < num_rows; i++){
        if(x[i] != *(xCopy + i)){
            printf("WRONG RES ON GPU on x[i] for i = %d\n", i); 
            break;
            return 1;
        }
    }

    // Print time
    printf("SYMGS Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SYMGS Time GPU: %.10lf\n", end_gpu - start_gpu);

    // Free
    free(row_ptr);
    free(col_ind);
    free(values);
    free(matrixDiagonal);

    CHECK(hipFree(dev_row_ptr));
    CHECK(hipFree(dev_col_ind));
    CHECK(hipFree(dev_values));
    CHECK(hipFree(dev_x));
    CHECK(hipFree(dev_matrixDiagonal));
    CHECK(hipFree(dev_x2));
    CHECK(hipFree(dev_locks));
    CHECK(hipFree(dev_changed));

    return 0;
}